#include "hip/hip_runtime.h"
/*
		EC527 Final Project --> FFT CUDA implementation
		Adin Horovitz, Monil Jhaveri, Evan Bowman

		Serial code is a slowed-down version of example found at:
			https://equilibriumofnothing.wordpress.com/2013/10/14/algorithm-iterative-fft/

		CUDA implementation uses example as a guide --> all STL functionality implemented using
		structs and direct complex value calculation :(

		to compile:

		nvcc -o FFT FFT_CUDA.cu

*/

#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <climits>
#include <cmath>
#include <complex>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <time.h> 
#include <stdint.h>
#include <thrust/complex.h>


#define N 1048576
#define MAX_THREAD 512
#define GIG 1000000000


typedef std::complex<double> cpx;
typedef std::vector<cpx> cVec;


/*		timing struct	*/
struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}


/* 		-- Complex Struct for CUDA --		*/
typedef struct compDouble{
	double real;
	double imag;
} d_cpx;


/* Forward Declare Serial Functions */
uint32_t reverseBits(uint32_t i);
int lg(uint32_t i);
int pown(const int p);
int find_absP(const int P);
int is_inverse(bool inverse);

void iterativeFFT(const cVec & primal, cVec & dual,const int P);
void run_CUDA(d_cpx * d_in, d_cpx * d_out, int P, bool inverse);

void init_vec(cVec &data);
void init_dVec(d_cpx * data);
void zero_dVec(d_cpx * data);
void print_dVec(d_cpx * data);
void print_cVec(cVec &data);
void print_both(cVec& cpu, d_cpx * gpu);
int compare_out(cVec& cpu, d_cpx * gpu, double prec);


// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}



/*		---		CUDA KERNELS		---		*/

__global__ void kernel_butterfly(d_cpx * input, d_cpx * output, int pwr2)
{
	// Identify indices to be swapped
	uint32_t thisIdx;
	uint32_t thatIdx;
//	int numBlocks = N/MAX_THREAD;

	if ( N <= MAX_THREAD)
		thisIdx = (uint32_t)threadIdx.x;
	else
		thisIdx = (MAX_THREAD * blockIdx.x) + threadIdx.x; 

	thatIdx = thisIdx;


	// SWAR to reverse bits of thisIdx
	register uint32_t mask = 0x55555555; // 0101...
	thatIdx = ((thatIdx & mask) << 1) | ((thatIdx >> 1) & mask);
	mask = 0x33333333; // 0011...
	thatIdx = ((thatIdx & mask) << 2) | ((thatIdx >> 2) & mask);
	mask = 0x0f0f0f0f; // 00001111...
	thatIdx = ((thatIdx & mask) << 4) | ((thatIdx >> 4) & mask);
	mask = 0x00ff00ff; // 0000000011111111...
	thatIdx = ((thatIdx & mask) << 8) | ((thatIdx >> 8) & mask);
	// 00000000000000001111111111111111 no need for mask
	thatIdx = (thatIdx << 16) | (thatIdx >> 16);

	thatIdx = thatIdx >> (32 - pwr2);

	//	printf("thisInd: %d		thatInd: %d\n", thisIdx, thatIdx);


	// Swap values
	output[thisIdx] = input[thatIdx];
	output[thatIdx] = input[thisIdx];

}


__global__ void kernel_FFTstage(d_cpx * input, int uStep, d_cpx uRoot, bool inverse)
{

	int numK = uStep / 2;

	// Allocate space in shared memory
//	__shared__ d_cpx[2 * MAX_THREAD];


	// Block position determines offset and k --> see IterativeFFT for parallel
	int offset = ((blockIdx.x * MAX_THREAD + threadIdx.x) / numK) * uStep;  
	int k = (blockIdx.x * MAX_THREAD + threadIdx.x) % numK;		

//	printf("[%d,%d] offset: %d, k: %d\n",threadIdx.x, threadIdx.y, offset, k);

	// omega = uRoot ^ k
	d_cpx omega, temp;
	omega.real = 1; omega.imag = 0;
	for(int i = 0; i < k; i++)
	{
		temp.real = (omega.real*uRoot.real) - (omega.imag*uRoot.imag);
		temp.imag = (omega.real*uRoot.imag) + (omega.imag*uRoot.real);
	
		omega = temp;
	}

//	printf("[%d,%d] uRoot: (%.3f,%.3f) omega: (%.3f,%.3f)\n", offset, k, uRoot.real, uRoot.imag, omega.real, omega.imag);


	// FFT stage change to input:

	int thisInd = offset + k;
	int thatInd = offset + k + uStep/2;

	temp = input[thatInd];
	d_cpx u = input[thisInd];

//	printf("thr[%d,%d] off=%d k=%d this:%d=(%.3f,%.3f) that:%d=(%.3f,%.3f)\n",blockIdx.x ,threadIdx.x, offset, k, 
//			thisInd, u.real, u.imag, thatInd, temp.real, temp.imag);



	// t = omega * temp
	d_cpx t;
	t.real = (omega.real*temp.real) - (omega.imag*temp.imag);
	t.imag = (omega.real*temp.imag) + (omega.imag*temp.real);

//	printf("[%d,%d] u: (%.3f,%.3f) t: (%.3f,%.3f)\n",offset, 
//			k, u.real, u.imag, t.real, t.imag);


	// Reuse temp for adding u + t
	temp.real = u.real + t.real;
	temp.imag = u.imag + t.imag;
	input[thisInd] = temp;

	// Reuse temp for subtracting u - t
	temp.real = u.real - t.real;
	temp.imag = u.imag - t.imag;
	input[thatInd] = temp;


	// If inverse FFT AND this is the last stage, divide through by N
	if (inverse && uStep == N/2)
	{
		input[thisInd].real /= N;
		input[thisInd].imag /= N;

		input[thatInd].real /= N;
		input[thatInd].imag /= N;
	}
}

/*		Globals		*/
int unityArray[2]; //stores unity step



/******************************************************************
**********                 BEGIN MAIN                   ***********
******************************************************************/
int main(int argc, char *argv[])
{

	printf("**************************************************\n");
	printf("*******     FFT CUDA Implementation       ********\n");
	printf("*******            N = %d                ********\n", N);
	printf("**************************************************\n");

	// Timing Structs
	struct timespec time1, time2, diffTime;
	struct timespec diff(struct timespec start, struct timespec end);  
	int clock_gettime(clockid_t clk_id, struct timespec *tp);
	hipEvent_t start, stop;	
	float elapsed_gpu;

	// easy case - assume N is even power of 2
	const int P = lg(N);

	// check to be sure N is pwr of 2
	if (N != pown(P)) {
	std::cout << "error, " << N << " is not an even power of 2" << std::endl;
		exit(1);
	}


	/*	---		Begin Serial (slow) execution	--- */


	// Serial Vectors can use STL
	cVec ser_in(N, 0);
	cVec ser_out(N, 0);
	init_vec(ser_in); // initialize vector for serial implementation


	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
	iterativeFFT(ser_in, ser_out, P); 
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);


/*
	// need another array for inverse
	cVec dualPrime(N, 0);

	// use -P as flag for inverse
	iterativeFFT(ser_out, ser_in, -P); 

*/
	//Timing output
	diffTime = diff(time1, time2); 




	/* ---		 Begin CUDA FFT			 	--- */


	// kernel complex numbers/comp arrays must use structs
	CUDA_SAFE_CALL(hipSetDevice(1));
	size_t allocSize = N * sizeof(d_cpx);
	d_cpx * h_in = (d_cpx *)malloc(allocSize);
	d_cpx * h_out = (d_cpx *)malloc(allocSize);

	// initialize device "vector"
	init_dVec(h_in);	
	zero_dVec(h_out);

	// Allocate space on Device
	d_cpx * d_in, *d_out;	// pointer to Device Memory
	CUDA_SAFE_CALL(hipMalloc((void **)&d_in, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_out, allocSize));


	// Transfer input array to device
	CUDA_SAFE_CALL(hipMemcpy(d_in, h_in, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_out, h_out, allocSize, hipMemcpyHostToDevice));


	// Start CUDA timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	run_CUDA(d_in, d_out, P, false);

	// Stop and destroy the timer
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

/*
	// Perform Inverse FFT:  YOU MUST COMMENT MEMCOPY FROM D_OUT IF REVERSING!!!
  run_CUDA(d_out, d_in, P, true);
	CUDA_SAFE_CALL(hipMemcpy(h_out, d_in, allocSize, hipMemcpyDeviceToHost));
*/


	// Transfer the results back to the host
	CUDA_SAFE_CALL(hipMemcpy(h_out, d_out, allocSize, hipMemcpyDeviceToHost));

//	print_dVec(h_out);


	/************** Print Outputs *************/
//	printf("\tCPU\t\t\tGPU\n");

	/**** Free Memory ****/
	CUDA_SAFE_CALL(hipFree(d_in));
	CUDA_SAFE_CALL(hipFree(d_out));
	free(h_in);





	/*****  Print timing  ******/
	//      printf("%ld", (long int)((double)(CPG)*(double)
	//		 (GIG * time_stamp[j][i].tv_sec + time_stamp[j][i].tv_nsec)));

	double time_CPU = (double)((GIG * diffTime.tv_sec + diffTime.tv_nsec) / 10e6);

//	printf("\n\n\n  CPU time: %ld.%.9ld (msec)\n", (long long)(diffTime.tv_sec * 1000), diffTime.tv_nsec * 1000); 
//	printf("\n\n\n  CPU time: %.6f (msec)\n", time_CPU); 
	printf("\n\n\nCPU time(msec)\tGPU time(msec)\n"); 
	printf("%.5f\t%.5f\n", time_CPU, elapsed_gpu);
	printf("  #Errors: %d\n\n\n\n",   compare_out(ser_out, h_out, 0.0001));

	return 0;
}



/****************************************************************
						END MAIN
****************************************************************/


void run_CUDA(d_cpx * d_in, d_cpx * d_out, int P, bool inverse)
{

/*
#define N 16
#define MAX_THREAD 8
#define NUM_BLOCKS N/MAX_BLOCK
#define THREADS_PER_BLOCK N/NUM_BLOCKS
*/

	int num_blocks = N / MAX_THREAD;
	if (num_blocks < 1)
		num_blocks = 1;
				
	dim3 reordGrid(num_blocks);
	dim3 reordBlock(N / num_blocks);


  printf("N: %d, #blocks: %d, TPB: %d\n", N, num_blocks, N/num_blocks);

	// Call Reorder Kernel
	kernel_butterfly<<< reordGrid, reordBlock>>>(d_in, d_out, P);



	/* -- FFT_Stage only requires N/2 total threads -- */
	/* -- Call 1/2 as many blocks as butterfly kernel -- */

	int FFT_blocks;
	int threads_per_block;

	if (num_blocks <= 1)
	{
		FFT_blocks = 1;
		threads_per_block = N / 2; 
		if (threads_per_block > MAX_THREAD)
		{
			printf("threads_per_block cannot exceed %d\n", MAX_THREAD);
			exit(0);
		}
	}
	else
	{
		FFT_blocks = num_blocks / 2;
		threads_per_block = MAX_THREAD;		
	}

	dim3 FFT_grid(FFT_blocks);
	dim3 dimBlock(threads_per_block);

	printf("#blocks: %d, TPB: %d\n", FFT_blocks, threads_per_block);

	int uStep;
	double theta;
	d_cpx uRoot;


	// Execute FFT P (lg(N)) times
	for (int i = 1; i <= P; i++)
	{

		// Update stage parameters
		uStep = 0x1 << i;
		theta = (inverse ? -2 : 2) * M_PI / uStep;
		uRoot.real = cos(theta);
		uRoot.imag = sin(theta);


//		printf("\nbegin Stage %d\n", i);

//		printf("theta: %.3f\n", theta);

		// Call FFT stage kernel
		kernel_FFTstage<<< FFT_grid, dimBlock >>>(d_out, uStep , uRoot, inverse);


		// Stage must complete before next begins
		hipDeviceSynchronize();

//		printf("end FFT Stage %d\n", i);
	}
}



void print_cVec(cVec & data)
{
	for(int i = 0; i < N; i++)
		printf("[%d]: (%.3f, %.3f)\n", i, std::real(data[i]), std::imag(data[i]));
}


void print_dVec(d_cpx * data)
{
	for(int i = 0; i < N; i++)
		printf("[%d]: (%.3f, %.3f)\n", i, data[i].real, data[i].imag);
}

void print_both(cVec& cpu, d_cpx * gpu)
{

  for(int i = 0; i < N; i++)
  {
	  printf("[%d]: (%.3f, %.3f)\t\t(%.3f, %.3f)\n", i, real(cpu[i]), imag(cpu[i]),
				gpu[i].real, gpu[i].imag);
	}
}


int compare_out(cVec& cpu, d_cpx * gpu, double prec)
{
	int errors = 0;

  for(int i = 0; i < N; i++)
  {
		
		if( abs(real(cpu[i]) - gpu[i].real) > prec  || abs(imag(cpu[i]) - gpu[i].imag) > prec)
			errors++;
	}
	
	return errors;
}


// Initialize Vector for serial implementation
void init_vec(cVec & data)
{
	for (int i = 0; i < N; i++)
		data[i] = i;
}


// initialize Vector for CUDA implementation
void init_dVec(d_cpx * data)
{
	for (int i = 0; i < N; i++)
	{
		data[i].real = i;
		data[i].imag = 0;
	}
}

// Zero CUDA array
void zero_dVec(d_cpx * data)
{
	for (int i = 0; i < N; i++)
	{
		data[i].real = 0;
		data[i].imag = 0;
	}
}


/*	Reorder Input values in Butterfly order */
uint32_t reverseBits(uint32_t i) {
  register uint32_t mask = 0x55555555; // 0101...
  i = ((i & mask) << 1) | ((i >> 1) & mask);
  mask = 0x33333333; // 0011...
  i = ((i & mask) << 2) | ((i >> 2) & mask);
  mask = 0x0f0f0f0f; // 00001111...
  i = ((i & mask) << 4) | ((i >> 4) & mask);
  mask = 0x00ff00ff; // 0000000011111111...
  i = ((i & mask) << 8) | ((i >> 8) & mask);
  // 00000000000000001111111111111111 no need for mask
  i = (i << 16) | (i >> 16);
  return i;
}

int lg(uint32_t i) {
  int count = -1;
  while (i) {
    i = i >> 1;
    count++;
  }
  return count;
}

// Russian peasant algorithm
// Checks if input is pwr of two
int pown(const int p) {
  uint32_t w = p;
  w |= w >> 1;
  w |= w >> 2;
  w |= w >> 4;
  w |= w >> 8;
  w |= w >> 16;
  uint32_t mask = w & ~(w >> 1);

  int a = 1;
  while (mask) {
    a = a * a;
    if (mask & p)
      a *= 2;
    mask >>= 1;
  }

  return a;
}

//determines if the P is positive or negative
int find_absP(const int P){ 
	int temp = P; 
	if (temp < 0){
		return -temp; 
	} 	
	else return temp; 

}

 
int is_inverse(bool inverse){
	if (inverse){
		return -1;
	}
	else return 1;	
} 

// FFT takes complex input vector, container for complex output, and lg2(N) (will be negative if inverse)
void iterativeFFT(const cVec & primal, cVec & dual,const int P) 
{
//  const int N = primal.size();
  const bool inverse = P < 0;
  const int absP = find_absP(P);	// Whether positive or negative, absP is always positive

  // bottom level of iteration tree --> puts elements in butterfly order
  for (int i = 0; i < N; i++)
    dual[i] = primal[reverseBits(i) >> (32 - absP)];

  // there are absP levels above the bottom
  for (int p = 1; p <= find_absP(P); p++) {

    // complex root of unity
    const int unityStep = 0x1 << p;	// --> starts at two, doubles each iteration
    unityArray[1] = unityStep; 
    
 
    const double theta = (is_inverse(inverse)) * 2 * M_PI / unityArray[1]; // INVERSE
    const cpx unityRoot(cos(theta), sin(theta));

    // each higher level doubles the step size
    for (int offset = 0; offset < primal.size(); offset += unityArray[1]) {
      cpx omega = 1;

      // combine within a step segment (note only iterate over half step)
      for (int k = 0; k < unityArray[1]/2; k++) {
        cpx u = dual[offset + k];

        const cpx t = omega * dual[offset + k + unityStep/2];
        omega *= unityRoot;

        dual[offset + k] = u + t;
        dual[offset + k + unityArray[1]/2] = u - t;
      }
    }
  }

  if (inverse) // INVERSE
    for (int j = 0; j < primal.size(); j++)
      dual[j] /= N;
}

